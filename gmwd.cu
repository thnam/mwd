#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <sys/time.h>

extern "C"{
#include "algo.h"
#include "vector.h"
}

Vector * ReadWF(const char * filename);
long int getMicrotime();

#define HANDLE_ERROR( err ) ( HandleError( err, __FILE__, __LINE__ ) )
#define DATAMB(bytes)			(bytes/1024/1024)
#define DATABW(bytes,timems)	((float)bytes/(timems * 1.024*1024.0*1024.0))

static void HandleError( hipError_t err, const char *file, int line ) {
  if (err != hipSuccess) {
    printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
    exit( EXIT_FAILURE );
  }
}

__global__ void gpuAdd(double *a, double *b, double *c, uint32_t n) {
  uint32_t gId = blockIdx.x*blockDim.x+threadIdx.x; // global id
  if (gId < n)
    c[gId] = a[gId] + b[gId];
}

__global__ void gpuMultiply(double *a, double *b, double f, uint32_t n) {
  uint32_t gId = blockIdx.x*blockDim.x+threadIdx.x; // global id
  if (gId < n)
    b[gId] = f * a[gId];
}

__global__ void gpuMovingAverage(double *a, double *b, uint32_t window,
    uint32_t n) {
}

int main(int argc, char *argv[]) {
  long int start = getMicrotime();
  Vector * wf = ReadWF("samples/purdue_full_wf0.csv");
  long int stop = getMicrotime();
  printf("Reading time %ld usec = %ld ms\n", (stop - start), (stop - start)/1000);

  double f = 0.999993;
  uint32_t M = 6000;
  uint32_t L = 600;

  start = getMicrotime();
  Vector * mwd = MWD(wf, f, M, L);
  stop = getMicrotime();
  printf("CPU MWD time %ld usec = %ld ms\n", (stop - start), (stop - start)/1000);

  // GPU things
	hipEvent_t		time1, time2, time3, time4;
	float totalTime, tfrCPUtoGPU, tfrGPUtoCPU, kernelExecutionTime; // GPU code run times
	hipError_t		cudaStatus;
  hipDeviceProp_t	GPUprop;
  uint32_t nBytes = wf->size * sizeof(double);
  /* double * hostWf0 = wf->data; */
  double * devWf0;
  double * devMWD;
  double * hostMWD = (double *) malloc(nBytes);
  char SupportedBlocks[100];

  int blockSize, gridSize;
  blockSize = 1024;
  gridSize = (int) ceil((float)nBytes / blockSize);
  printf("blockSize %d, gridSize %d\n", blockSize, gridSize);

  hipEventCreate(&time1);
  hipEventCreate(&time2);
  hipEventCreate(&time3);
  hipEventCreate(&time4);

  hipEventRecord(time1, 0);
	int NumGPUs = 0;
	hipGetDeviceCount(&NumGPUs);
	if (NumGPUs == 0){
		printf("\nNo CUDA Device is available\n");
		exit(EXIT_FAILURE);
	}
	cudaStatus = hipSetDevice(0);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		exit(EXIT_FAILURE);
	}
	hipGetDeviceProperties(&GPUprop, 0);
	uint32_t SupportedKBlocks = (uint32_t)GPUprop.maxGridSize[0]
    * (uint32_t)GPUprop.maxGridSize[1] * (uint32_t)GPUprop.maxGridSize[2] / 1024;
	uint32_t SupportedMBlocks = SupportedKBlocks / 1024;
	sprintf(SupportedBlocks, "%u %c",
      (SupportedMBlocks >= 5) ? SupportedMBlocks : SupportedKBlocks,
      (SupportedMBlocks >= 5) ? 'M' : 'K');
	uint32_t MaxThrPerBlk = (uint32_t)GPUprop.maxThreadsPerBlock;

	printf("--------------------------------------------------------------------------\n");
	printf("%s    ComputeCapab=%d.%d  [max %s blocks; %d thr/blk] \n", 
			GPUprop.name, GPUprop.major, GPUprop.minor, SupportedBlocks, MaxThrPerBlk);
	printf("--------------------------------------------------------------------------\n");

  HANDLE_ERROR(hipMalloc((void **) &devWf0, nBytes));
  HANDLE_ERROR(hipMalloc((void **) &devMWD, nBytes));

	HANDLE_ERROR(hipMemcpy(devWf0, wf->data, nBytes, hipMemcpyHostToDevice));

	hipEventRecord(time2, 0);		// Time stamp after the CPU --> GPU tfr is done
  gpuMultiply<<<gridSize, blockSize>>>(devWf0, devMWD, f, wf->size);
  cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr,
        "\ncudaDeviceSynchronize returned error code %d after launching the kernel!\n", cudaStatus);
		exit(EXIT_FAILURE);
	}

	hipEventRecord(time3, 0);
  HANDLE_ERROR(hipMemcpy(hostMWD, devMWD, nBytes, hipMemcpyDeviceToHost));
  hipEventRecord(time4, 0);

	hipEventSynchronize(time1);
	hipEventSynchronize(time2);
	hipEventSynchronize(time3);
	hipEventSynchronize(time4);
	hipEventElapsedTime(&totalTime, time1, time4);
	hipEventElapsedTime(&tfrCPUtoGPU, time1, time2);
	hipEventElapsedTime(&kernelExecutionTime, time2, time3);
	hipEventElapsedTime(&tfrGPUtoCPU, time3, time4);

	cudaStatus = hipDeviceSynchronize();
  /* checkError(hipGetLastError());	// screen for errors in kernel launches */
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "\n Program failed after hipDeviceSynchronize()!");
		exit(EXIT_FAILURE);
	}

  uint32_t i = 0;
  for (i = 0; i < wf->size; ++i) {
    printf("%lf %lf\n", wf->data[i], hostMWD[i]);
  }

	printf("CPU->GPU Transfer   =%7.2f ms  ...  %4d MB  ...  %6.2f GB/s\n",
      tfrCPUtoGPU, DATAMB(nBytes), DATABW(nBytes, tfrCPUtoGPU));
	printf("Kernel Execution    =%7.2f ms  ...  %4d MB  ...  %6.2f GB/s\n",
      kernelExecutionTime, DATAMB(2*nBytes), DATABW(2*nBytes, kernelExecutionTime));
	printf("GPU->CPU Transfer   =%7.2f ms  ...  %4d MB  ...  %6.2f GB/s\n",
      tfrGPUtoCPU, DATAMB(nBytes), DATABW(nBytes, tfrGPUtoCPU));
	printf("--------------------------------------------------------------------------\n");
	printf("Total time elapsed  =%7.2f ms       %4d MB  ...  %6.2f GB/s\n",
      totalTime, DATAMB((2 * nBytes + 2*nBytes)),
      DATABW((2 * nBytes + 2*nBytes), totalTime));
  printf("--------------------------------------------------------------------------\n\n");


  // done
  hipFree(devWf0);
  hipFree(devMWD);
  free(hostMWD);
  VectorFree(mwd);
  VectorFree(wf);
  return 0;
}

Vector * ReadWF(const char *filename){
  Vector * wf0 = VectorInit();
  FILE *fp = fopen(filename, "r");
  if (fp == NULL) {
    printf("Cannot open file.\n");
    exit(1);
  }

  double val0, val1;
  while (fscanf(fp, "%lf,%lf", &val0, &val1) == 2) {
    VectorAppend(wf0, val1);
  }

  fclose(fp);
  return wf0;
}

long int getMicrotime(){
  struct timeval currentTime;
  gettimeofday(&currentTime, NULL);
  return currentTime.tv_sec * (int)1e6 + currentTime.tv_usec;
}
