#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <sys/time.h>

extern "C"{
#include "algo.h"
#include "vector.h"
}

Vector * ReadWF(const char * filename);
long int getMicrotime();

#define HANDLE_ERROR( err ) ( HandleError( err, __FILE__, __LINE__ ) )

static void HandleError( hipError_t err, const char *file, int line ) {
  if (err != hipSuccess) {
    printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
    exit( EXIT_FAILURE );
  }
}

__global__ void vecAdd(double *a, double *b, double *c, int n) {
  // Get our global thread ID
  int id = blockIdx.x*blockDim.x+threadIdx.x;
  // Make sure we do not go out of bounds
  if (id < n)
    c[id] = a[id] + b[id];
}

__global__ void vecMultiply(double *a, double *b, double f, int n) {
  // Get our global thread ID
  int id = blockIdx.x*blockDim.x+threadIdx.x;
  // Make sure we do not go out of bounds
  if (id < n)
    b[id] = f * a[id];
}

int main(int argc, char *argv[]) {
  long int start = getMicrotime();
  Vector * wf = ReadWF("samples/purdue_full_wf0.csv");
  long int stop = getMicrotime();
  printf("Reading time %ld usec = %ld ms\n", (stop - start), (stop - start)/1000);

  start = getMicrotime();
  Vector * mwd = MWD(wf, 0.999993, 6000, 600);
  stop = getMicrotime();
  printf("CPU MWD time %ld usec = %ld ms\n", (stop - start), (stop - start)/1000);

  // GPU things
	hipEvent_t		time1, time2, time3, time4;
	hipError_t		cudaStatus, cudaStatus2;
	/* hipDeviceProp_t	GPUprop; */
  uint32_t nBytes = wf->size * sizeof(double);
  /* double * hostWf0 = wf->data; */
  double * devWf0;
  double * devMWD;
  double * hostMWD = (double *) malloc(nBytes);
  double f = 0.999993;

  int blockSize, gridSize;
  blockSize = 1024;
  gridSize = (int) ceil((float)wf->size / blockSize);

  hipEventCreate(&time1);
  hipEventCreate(&time2);
  hipEventCreate(&time3);
  hipEventCreate(&time4);

  hipEventRecord(time1, 0);
  cudaStatus = hipMalloc((void **) &devWf0, wf->size);
  cudaStatus2 = hipMalloc((void **) &devMWD, wf->size);
	if ((cudaStatus != hipSuccess) || (cudaStatus2 != hipSuccess)){
		fprintf(stderr, "hipMalloc failed! Can't allocate GPU memory");
		exit(EXIT_FAILURE);
	}

	cudaStatus = hipMemcpy(devWf0, wf->data, nBytes, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy  CPU to GPU  failed!");
		exit(EXIT_FAILURE);
	}

	hipEventRecord(time2, 0);		// Time stamp after the CPU --> GPU tfr is done
  vecMultiply<<<gridSize, blockSize>>>(devWf0, devMWD, f, wf->size);
  hipEventRecord(time3, 0);
  hipMemcpy(hostMWD, devMWD, nBytes, hipMemcpyDeviceToHost);
  hipEventRecord(time4, 0);

  uint32_t i = 0;
  for (i = 0; i < wf->size; ++i) {
    printf("%lf\n", devMWD[i]);
  }

  // done
  hipFree(devWf0);
  hipFree(devMWD);
  free(hostMWD);
  VectorFree(mwd);
  VectorFree(wf);
  return 0;
}

Vector * ReadWF(const char *filename){
  Vector * wf0 = VectorInit();
  FILE *fp = fopen(filename, "r");
  if (fp == NULL) {
    printf("Cannot open file.\n");
    exit(1);
  }

  double val0, val1;
  while (fscanf(fp, "%lf,%lf", &val0, &val1) == 2) {
    VectorAppend(wf0, val1);
  }

  fclose(fp);
  return wf0;
}

long int getMicrotime(){
  struct timeval currentTime;
  gettimeofday(&currentTime, NULL);
  return currentTime.tv_sec * (int)1e6 + currentTime.tv_usec;
}
